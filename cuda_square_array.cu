#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>


__global__ void square(float *d_out, float *d_in) {
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx]=f*f;
}






// Function to print an array on the host
void printArray(float *arr, int size) {
    for (int i = 0; i < size; ++i) {
        printf("%f ", arr[i]);
    }
    printf("\n");
}




int main(int argc, char **argv) {


  const int ARRAY_SIZE = 64;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);


  float h_in[ARRAY_SIZE];
  float h_out[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; i++) {
    h_in[i] = float(i);
  }




  float *d_in;
  float *d_out;
  hipMalloc((void **)&d_in, ARRAY_BYTES);
  hipMalloc((void **)&d_out, ARRAY_BYTES);


    //copy results to GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    //Launch the Kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);


    //copy results back from GPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);


    //release GPU memory
    hipFree(d_in);
    hipFree(d_out);




    printArray(h_out, ARRAY_SIZE);
}
