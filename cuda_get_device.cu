%%cuda
#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipDeviceProp_t properties;

    // Get the current device
    hipGetDevice(&device);
    // Get device properties
    hipGetDeviceProperties(&properties, device);

    printf("Device Name: %s\n", properties.name);
    printf("Compute Capability: %d.%d\n", properties.major, properties.minor);
    printf("Max Threads per Block: %d\n", properties.maxThreadsPerBlock);
    printf("Max Threads per Multiprocessor: %d\n", properties.maxThreadsPerMultiProcessor);
    printf("Max Blocks per Multiprocessor: %d\n", properties.maxBlocksPerMultiProcessor);
    printf("Warp Size: %d\n", properties.warpSize);
    printf("Number of SMs: %d\n", properties.multiProcessorCount);

    return 0;
}
